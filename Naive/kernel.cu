#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
	
}


hipError_t searchPattern(char *string,char*pat,int* res);

__global__ void searchPatternKernel(const char *string,const char*pat,int patlen,int segstringlen,int lastsegstringlen,int *res)
{
	int startIndex =threadIdx.x;
	printf("%d\n",startIndex);
	int pos;//the pos of this segment 

	//divide into 4 segment 0123
	pos = startIndex*segstringlen;
	printf("pos:%d\n",pos);
	/*if (string[i]==pat[0])
	{
		int j;
		for(j=1;j<4;j++)
		{ 
			if (string[i+j]!=pat[j])break;
			else res[i]=1;
		}
	}*/

	int strl;//the length of the segment
	if (startIndex<3)
	{
		strl=segstringlen+patlen;
	}
	else strl=lastsegstringlen;
	printf("%d\n",strl);
	
	printf("pati :%s %d\n",pat,startIndex);
	printf("string i:%s %d\n",string,startIndex);
	
	
	
	int i;
	for (i=pos;i<pos+strl-patlen+1;i++)
	{
	
	int flag=1;

		int j;
		for (j=0;j<patlen;j++)
		{
			if (pat[j]!=string[j+i]){flag=0;break;}
		}
			if(flag)res[i]=1;

	}

}



int main()
{
   /*const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
   hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);*/

	char string[]="wefdfewfjwfbhwyfghwqfbhweyhwefhwefewbfwfhbwuw";
    char pat[]="wef";
	int const  datasize= strlen(string);

	int* res;
	res=(int*)malloc((datasize)* sizeof(int));

	 memset(res, 0, datasize);
	 //searchPattern(char* string, char*pat)
	hipError_t cudaStatus = searchPattern(string, pat,res);
if (cudaStatus != hipSuccess) {
  fprintf(stderr, "searchPattern failed!");
  return 1;
}

// Print out the string match result position
int total_matches = 0;
for (int i=0; i<datasize; i++) {
  if (res[i] == 1) {
    printf("Character found at position % i\n", i);
    total_matches++;
  }
}
printf("Total matches = %d\n", total_matches);


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	system("pause");
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}


hipError_t searchPattern(char* string, char*pat,int *res)
{
    char *dev_string = 0;
    char *dev_pat = 0;
    int *dev_res = 0;
	int stringlen=strlen(string);
	int patlen=strlen(pat);
		printf("%d\n",patlen);
	
	int segstrlen=stringlen/4;
		printf("%d\n",segstrlen);
	int lastsegstrlen= segstrlen+stringlen%4;
		printf("%d\n",lastsegstrlen);

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_string,(stringlen)* sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
	
	cudaStatus = hipMalloc((void**)&dev_pat, (patlen) * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_res, (stringlen) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.

	cudaStatus = hipMemcpy(dev_string, string, (stringlen) * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMemcpy1 failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_pat, pat, (patlen)* sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMemcpy2 failed!");
        goto Error;
    }
	
	cudaStatus = hipMemset(dev_res, 0, (stringlen)* sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemset failed!");
        goto Error;
    }
	
    // Launch a kernel on the GPU with one thread for each element.
    //searchPatternKernel<<<1, size>>>(dev_res, dev_string, dev_pat);

	searchPatternKernel<<<1,4>>>(dev_string,dev_pat,patlen,segstrlen,lastsegstrlen,dev_res);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "PatternKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
   // cudaStatus = hipDeviceSynchronize();
	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching searchpatternKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(res, dev_res, (stringlen) * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
   hipFree(dev_res);
    hipFree(dev_pat);
    hipFree(dev_string);
    
    return cudaStatus;
}
