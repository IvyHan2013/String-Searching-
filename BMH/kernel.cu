#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctime>
#include <iostream>
#include <Windows.h>
using namespace std;
#define MAXCHAR 256

#define N	1024
#define THREADS_PER_BLOCK 16
_LARGE_INTEGER time_start;
_LARGE_INTEGER time_over;
double dqFreq;
hipError_t searchPattern(char *string,char*pat,int* res);

void makeBadCharTable(int* table, char* pat,int patlen)
{
    //init table with all char cannot be found
    int i;
    for (i=0; i<MAXCHAR;i++)
    {
        table[i]=patlen;//not found
    }
    for (i=0;i<patlen-1;i++)
    {
        table[pat[i]]=patlen-1-i;
    }

}

__global__ void searchPatternKernel(const char *string,const char*pat,int patlen,int segstringlen,int lastsegstringlen,int *res,int* table)
{
	int startIndex =threadIdx.x+blockIdx.x * blockDim.x;
	//printf("%d\n",startIndex);
	int pos;//the pos of this segment 

	//divide into nthread segment 
	pos = startIndex*segstringlen;
	
	

	int strl;//the length of the segment
	if (startIndex<N-1)
	{
		strl=segstringlen+patlen;
	}
	else strl=lastsegstringlen;
	

	int i;

	/*__shared__ char substring[49151];

	for (i=0;i<strl;i++)
	{
		substring[i]=string[i+pos];
	}*/

	for (i=pos;i<pos+strl-patlen+1;)
	{
	
		int j= patlen-1;

        while(j>=0&& pat[j]==string[i+j])
        {
            j--;
        }
        if (j<0)
        {
			res[i]=1;

        }

        i+=table[string[i+patlen-1]];
	}

	
	/*for (i=0;i<strl-patlen+1;)
	{
	
		int j= patlen-1;

        while(j>=0&& pat[j]==substring[i+j])
        {
            j--;
        }
        if (j<0)
        {
			res[i+pos]=1;

        }

        i+=table[substring[i+patlen-1]];
	}
	__syncthreads();*/

}


int main()
{

	char *string;
	char *pat;
	
	
	freopen("input6.txt", "r", stdin);


	/*char tmp;
	int ll=0;
	tmp=cin.get();

	while (tmp!='\n')
	{
		ll++;
		tmp=cin.get();
	}

	printf("%d\n",ll);
	tmp=cin.get();
	ll=0;
	while (tmp!='\n')
	{
		ll++;
		tmp=cin.get();
	}
	printf("%d\n",ll);
	return 0;*/



	string = new char[400000000];
	pat = new char[40000];

	cin.getline(string, 400000000, '\n');
	cin.getline(pat, 40000, '\n');


	
	//char string[]="wefdfewfjwfbhwyfghwqfbhweyhwefhwefewbfwfhbwuw";
    //char pat[]="wef";
	int const  datasize= strlen(string);

	int* res;
	res=(int*)malloc((datasize)* sizeof(int));

	memset(res, 0, datasize);

	 //searchPattern(char* string, char*pat)


	 clock_t start_time=clock();

    
        hipError_t cudaStatus = searchPattern(string, pat,res);
     

     clock_t end_time=clock();
     
	
	

	
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "searchPattern failed!");
		return 1;
	}

	// Print out the string match result position
	int total_matches = 0;
	for (int i=0; i<datasize; i++) {
	  if (res[i] == 1) {
		printf("Character found at position % i\n", i);
		total_matches++;
	  }
	}
	printf("Total matches = %d\n", total_matches);
	//printf("time cost: %fms\n", 1000 * ((time_over.QuadPart - time_start.QuadPart) / dqFreq));
	 cout<< "Running time is: "<<static_cast<double>(end_time-start_time)/CLOCKS_PER_SEC*1000<<"ms"<<endl;//�������ʱ��
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	system("pause");
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.


hipError_t searchPattern(char* string, char*pat,int *res)
{
    char *dev_string = 0;
    char *dev_pat = 0;
    int *dev_res = 0;
	int stringlen=strlen(string);
	int patlen=strlen(pat);
	//	printf("%d\n",patlen);
	
	int segstrlen=stringlen/N;
		//printf("%d\n",segstrlen);

	int lastsegstrlen= segstrlen+stringlen%N;
		//printf("%d\n",lastsegstrlen);


	
	 int table[MAXCHAR];
	
    makeBadCharTable(table,pat,patlen);

	int *dev_table=0;

    hipError_t cudaStatus;
	 

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_string,(stringlen)* sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
	
	cudaStatus = hipMalloc((void**)&dev_pat, (patlen) * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_res, (stringlen) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_table, (MAXCHAR) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.

	cudaStatus = hipMemcpy(dev_string, string, (stringlen) * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMemcpy1 failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_pat, pat, (patlen)* sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMemcpy2 failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_table, table, (MAXCHAR)* sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMemcpy2 failed!");
        goto Error;
    }
	
	cudaStatus = hipMemset(dev_res, 0, (stringlen)* sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemset failed!");
        goto Error;
    }
		LARGE_INTEGER f;
	QueryPerformanceFrequency(&f);
	dqFreq = (double)f.QuadPart;
    // Launch a kernel on the GPU with one thread for each element.
   
	QueryPerformanceCounter(&time_start);

	
	searchPatternKernel<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(dev_string,dev_pat,patlen,segstrlen,lastsegstrlen,dev_res,dev_table);
	
	
	QueryPerformanceCounter(&time_over);


	


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "PatternKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
   // cudaStatus = hipDeviceSynchronize();
	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching searchpatternKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(res, dev_res, (stringlen) * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
   hipFree(dev_res);
    hipFree(dev_pat);
    hipFree(dev_string);
    
    return cudaStatus;
}
